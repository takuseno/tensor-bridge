#include "native_tensor_bridge.hpp"

void native_copy_tensor(const DataPtr& dst_ptr, DataPtr& src_ptr) {
    hipSetDevice(src_ptr.device);
    if (src_ptr.device != dst_ptr.device) {
        hipDeviceEnablePeerAccess(dst_ptr.device, 0);
    }
    hipMemcpy((void*) dst_ptr.ptr, (const void*) src_ptr.ptr, src_ptr.size, hipMemcpyDeviceToDevice);
}
