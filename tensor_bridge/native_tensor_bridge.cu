#include "native_tensor_bridge.hpp"

void native_copy_tensor(const DataPtr& src_ptr, DataPtr& dst_ptr) {
    hipSetDevice(src_ptr.device);
    if (src_ptr.device != dst_ptr.device) {
        hipDeviceEnablePeerAccess(dst_ptr.device, 0);
    }
    hipMemcpy((void*) src_ptr.ptr, (const void*) dst_ptr.ptr, src_ptr.size, hipMemcpyDeviceToDevice);
}
